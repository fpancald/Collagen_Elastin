#include "functor_advance_pos.h"

#include "system_structures.h"
#include "system.h"
#include "advance_positions.h"


void advance_positions(
	NodeInfoVecs& nodeInfoVecs,
	GeneralParams& generalParams,
	EdgeInfoVecs& edgeInfoVecs,
	RandVecs& randVecs) {


		//At this point, the previous node location is the same as the current node,
		//we can therefore use previous node locations to update nodeLoc.
		unsigned _seed = rand();
    	thrust::counting_iterator<unsigned> index_sequence_begin(_seed);

    	thrust::transform(thrust::device, index_sequence_begin, index_sequence_begin + (generalParams.max_node_count),
        	randVecs.gaussianData.begin(), psrunifgen(-1.0, 1.0));

		thrust::counting_iterator<unsigned> nodeIndexBegin(0);

		thrust::transform(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeIndexBegin,
					nodeInfoVecs.node_loc_x.begin(),
					nodeInfoVecs.node_loc_y.begin(),
					nodeInfoVecs.node_loc_z.begin(),
					nodeInfoVecs.node_upper_selection_pull.begin(),
					nodeInfoVecs.node_lower_selection_pull.begin())),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeIndexBegin,
					nodeInfoVecs.node_loc_x.begin(),
					nodeInfoVecs.node_loc_y.begin(),
					nodeInfoVecs.node_loc_z.begin(),
					nodeInfoVecs.node_upper_selection_pull.begin(),
					nodeInfoVecs.node_lower_selection_pull.begin())) + generalParams.max_node_count,
			//second vector begin
			thrust::make_zip_iterator(
				thrust::make_tuple(
					randVecs.gaussianData.begin(),
					nodeInfoVecs.node_force_x.begin(),
					nodeInfoVecs.node_force_y.begin(),
					nodeInfoVecs.node_force_z.begin())),
			//save result in third vector to test values
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_loc_x.begin(),
					nodeInfoVecs.node_loc_y.begin(),
					nodeInfoVecs.node_loc_z.begin(),
					nodeInfoVecs.node_vel.begin())),
			functor_advance_pos(
				generalParams.dt,
				edgeInfoVecs.viscosity_collagen,
				edgeInfoVecs.viscosity_elastin,
				edgeInfoVecs.temperature,
				edgeInfoVecs.kB,
				edgeInfoVecs.node_mass,
				generalParams.max_node_count,
				thrust::raw_pointer_cast(nodeInfoVecs.node_is_collagen.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_is_elastin.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.is_node_fixed.data())));

}
