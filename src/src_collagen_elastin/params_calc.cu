#include "system_structures.h"
#include "functor_misc.h"

#include "system.h"
#include "params_calc.h"
#include "functor_calc_strain_params.h"


void params_calc(
    EdgeInfoVecs& edgeInfoVecs,
    NodeInfoVecs& nodeInfoVecs,
    GeneralParams& generalParams,
    PltInfoVecs& pltInfoVecs) {

		//count positive and negative strains for edges that are not added. If an edge is added, a zero is placed on that strain.
		//notice that each thread will count edges twice, so divide by two at the end
	
		thrust::fill(nodeInfoVecs.discretized_edges_strain.begin(), nodeInfoVecs.discretized_edges_strain.end(),0.0);
		thrust::fill(nodeInfoVecs.discretized_edges_alignment.begin(), nodeInfoVecs.discretized_edges_alignment.end(),0.0);	

		//copy current host information to device for strain calculation. 
		thrust::copy(nodeInfoVecs.host_edge_left.begin(),
			nodeInfoVecs.host_edge_left.begin() + generalParams.current_edge_count,
			nodeInfoVecs.device_edge_left.begin());

		thrust::copy(nodeInfoVecs.host_edge_right.begin(),
			nodeInfoVecs.host_edge_right.begin() + generalParams.current_edge_count,
			nodeInfoVecs.device_edge_right.begin());

		thrust::transform(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.device_edge_left.begin(),
					nodeInfoVecs.device_edge_right.begin())),
					 
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.device_edge_left.begin(),
					nodeInfoVecs.device_edge_right.begin())) + generalParams.current_edge_count,
					
			//outputs discretized strain etc			
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.discretized_edges_strain.begin(),
					nodeInfoVecs.discretized_edges_alignment.begin())),
					
			functor_calc_strain_params(
				generalParams.origin_node_count,
				generalParams.max_node_count,
				generalParams.max_nbr_count,
				thrust::raw_pointer_cast(nodeInfoVecs.node_loc_x.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_loc_y.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_loc_z.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.num_origin_nbr_per_node_vec.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.current_node_edge_count_vec.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.global_neighbors.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.global_length_zero.data()) ));
		
			thrust::transform(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.node_force_x.begin(),
						nodeInfoVecs.node_force_y.begin(),
						nodeInfoVecs.node_force_z.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.node_force_x.begin(),
						nodeInfoVecs.node_force_y.begin(),
						nodeInfoVecs.node_force_z.begin())) + generalParams.max_node_count,
				nodeInfoVecs.sum_forces_on_node.begin(),//save vector
                functor_norm());

};