#include "hip/hip_runtime.h"
#include <thrust/system_error.h>
#include <thrust/binary_search.h>
#include <thrust/reduce.h>
#include <algorithm>
#include <thrust/replace.h>
#include <thrust/unique.h>
#include <thrust/gather.h>
#include <ostream>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <math.h>

#include "storage.h"
#include "system_builder.h"
#include "collagen_elastin_spring.h"
#include "bending_spring.h"
#include "advance_positions.h"
#include "bucket_scheme.h"
#include "link_nodes.h"
#include "external_force.h"
#include "system.h"
#include "functor_misc.h"

using namespace thrust::placeholders;

void System::set_bucket_scheme(){
	init_dim_general(nodeInfoVecs, domainParams, auxVecs, generalParams);
	init_net_inct_bucket(nodeInfoVecs, domainParams, auxVecs, generalParams);
	build_net_inct_bucket(nodeInfoVecs, domainParams, auxVecs, generalParams);
	extend_net_inct_bucket(nodeInfoVecs, domainParams, auxVecs, generalParams);
}

void System::solve_forces() {

	thrust::fill(nodeInfoVecs.node_force_x.begin(),nodeInfoVecs.node_force_x.end(),0);
	thrust::fill(nodeInfoVecs.node_force_y.begin(),nodeInfoVecs.node_force_y.end(),0);
	thrust::fill(nodeInfoVecs.node_force_z.begin(),nodeInfoVecs.node_force_z.end(),0);
	
	double addedLinks = generalParams.current_edge_count - generalParams.origin_edge_count;

	if (generalParams.linking == true) {

		link_nodes(nodeInfoVecs, edgeInfoVecs, auxVecs, generalParams);
	}

	//apply external force.
	/*external_force(
		nodeInfoVecs,
		generalParams,
		extensionParams,
		domainParams);*/

	//only counts external force on network nodes since force has been reset.



  	calc_bending_spring_force(nodeInfoVecs, bendInfoVecs, generalParams);
	calc_spring_force(nodeInfoVecs, edgeInfoVecs, generalParams);
	extensionParams.totalAppliedForce = thrust::transform_reduce(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.node_force_x.begin(),
				nodeInfoVecs.node_force_y.begin(),
				nodeInfoVecs.node_force_z.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.node_force_x.begin(),
				nodeInfoVecs.node_force_y.begin(),
				nodeInfoVecs.node_force_z.begin())) + generalParams.max_node_count,
			functor_norm(), 0.0, thrust::plus<double>() );
	
	//std::cout<<" total applied force: " << extensionParams.totalAppliedForce << std::endl;
};


void System::solve_system() {

	double lastTime = 0.0;
	bool runIters = true;
	set_bucket_scheme();

	while (runIters == true) {

		generalParams.iterationCounter++;
		generalParams.currentTime += generalParams.dt;
		//std::cout << "current iter: " <<generalParams.iterationCounter<<  std::endl;
		//if (generalParams.iterationCounter % 50 == 0){
		set_bucket_scheme();
		//}

		advance_positions(
			nodeInfoVecs,
			generalParams,
			edgeInfoVecs,
      		randVecs);


		if ((generalParams.iterationCounter % 20000) == 0) {
			//storage->print_VTK_file();
		}

		solve_forces(); //resets and solves forces for next time step

		if ((generalParams.iterationCounter % 20000) == 0) {
			double currentStrain = (extensionParams.averageUpperStrain - extensionParams.averageLowerStrain) /
			(extensionParams.originAverageUpperStrain - extensionParams.originAverageLowerStrain ) - 1.0;
			if (currentStrain>4.0){
				runIters=false;
			}
			thrust::transform(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.node_force_x.begin(),
						nodeInfoVecs.node_force_y.begin(),
						nodeInfoVecs.node_force_z.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.node_force_x.begin(),
						nodeInfoVecs.node_force_y.begin(),
						nodeInfoVecs.node_force_z.begin())) + generalParams.max_node_count,
				nodeInfoVecs.sum_forces_on_node.begin(),//save vector
				functor_norm());

			storage->updateTotalStrain();
		}


		double maxVel = *(thrust::max_element(nodeInfoVecs.node_vel.begin(), nodeInfoVecs.node_vel.end()));

		thrust::device_vector<double>::iterator iter = thrust::max_element(nodeInfoVecs.node_vel.begin(), nodeInfoVecs.node_vel.end());

		unsigned position = iter - nodeInfoVecs.node_vel.begin();
		double max_val = *iter;

		//std::cout << "The maximum value is " << max_val << " at position " << position << std::endl;
		//std::cout<< "node :" << nodeInfoVecs.node_loc_x[0] << " " << nodeInfoVecs.node_loc_y[0] << " " << nodeInfoVecs.node_loc_z[0] << std::endl;
		
		/*
		if (maxVel < generalParams.epsilon) {
			//store sum of all forces on each node. Used in stress calculations
			thrust::transform(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.node_force_x.begin(),
						nodeInfoVecs.node_force_y.begin(),
						nodeInfoVecs.node_force_z.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.node_force_x.begin(),
						nodeInfoVecs.node_force_y.begin(),
						nodeInfoVecs.node_force_z.begin())) + generalParams.max_node_count,
				nodeInfoVecs.sum_forces_on_node.begin(),//save vector
				functor_norm());


			generalParams.epsilon = (1.0) *
				sqrt(6.0 * edgeInfoVecs.kB * edgeInfoVecs.temperature * generalParams.dt / edgeInfoVecs.viscosity_collagen);

			std::cout<<"Maximum vel: "<< maxVel <<std::endl;
			std::cout<<"updating epsilon back to original: "<< generalParams.epsilon<<std::endl;
			generalParams.magnitudeForce += generalParams.df;
			std::cout<<"magnitudeForce: "<< generalParams.magnitudeForce<<std::endl;

		}*/
		///////////////////////////////////////////////////////////////////////////////
		//EQUILIBRIUM END
		//////////////////////////////////////////////////////////////////////

	}

};

System::System()  {};

void System::assign_storage(std::shared_ptr<Storage> _storage) {
	storage = _storage;
}

void System::initialize_system(HostNodeInfoVecs& hostNodeInfoVecs) {

	std::cout<< "total Edge Count: "<< generalParams.origin_edge_count << std::endl;
	std::cout << "max num nodes: " << generalParams.max_node_count << std::endl;

	nodeInfoVecs.origin_edge_left = hostNodeInfoVecs.host_spring_edge_left;
	nodeInfoVecs.origin_edge_right = hostNodeInfoVecs.host_spring_edge_right;

	set_node_vecs(//calls initDimensionBucketScheme
		hostNodeInfoVecs);

	set_bend_vecs(hostNodeInfoVecs);

	set_edge_vecs(hostNodeInfoVecs);

	set_extras();
};


void System::set_node_vecs(
	HostNodeInfoVecs& hostNodeInfoVecs) {

	randVecs.gaussianData.resize(generalParams.max_node_count);

	nodeInfoVecs.id_edges_made_temp.resize(generalParams.max_node_count * generalParams.max_links_per_iteration);//corresponds to upperAdj vector size plus a single value to hold number of added nodes
	thrust::fill(nodeInfoVecs.id_edges_made_temp.begin(), nodeInfoVecs.id_edges_made_temp.end(), 0);

	nodeInfoVecs.sum_forces_on_node.resize(generalParams.max_node_count);

	nodeInfoVecs.node_upper_selection_pull.resize(generalParams.max_node_count);
	nodeInfoVecs.node_lower_selection_pull.resize(generalParams.max_node_count);

	nodeInfoVecs.node_vel.resize(generalParams.max_node_count);

  	nodeInfoVecs.node_is_collagen.resize(generalParams.max_node_count);
  	nodeInfoVecs.node_is_elastin.resize(generalParams.max_node_count);

	nodeInfoVecs.node_loc_x.resize(generalParams.max_node_count);
	nodeInfoVecs.node_loc_y.resize(generalParams.max_node_count);
	nodeInfoVecs.node_loc_z.resize(generalParams.max_node_count);
	nodeInfoVecs.node_vel_x.resize(generalParams.max_node_count);
	nodeInfoVecs.node_vel_y.resize(generalParams.max_node_count);
	nodeInfoVecs.node_vel_z.resize(generalParams.max_node_count);


	nodeInfoVecs.node_force_x.resize(generalParams.max_node_count);
	nodeInfoVecs.node_force_y.resize(generalParams.max_node_count);
	nodeInfoVecs.node_force_z.resize(generalParams.max_node_count);

	nodeInfoVecs.discretized_edges_strain.resize(generalParams.max_node_count * generalParams.max_nbr_count);
	nodeInfoVecs.discretized_edges_alignment.resize(generalParams.max_node_count * generalParams.max_nbr_count);

	//sized larger for input later
	
	nodeInfoVecs.device_edge_left.resize(generalParams.max_node_count * generalParams.max_nbr_count);
	nodeInfoVecs.device_edge_right.resize(generalParams.max_node_count * generalParams.max_nbr_count);

	nodeInfoVecs.host_edge_left.resize(generalParams.max_node_count * generalParams.max_nbr_count);
	nodeInfoVecs.host_edge_right.resize(generalParams.max_node_count * generalParams.max_nbr_count);


	thrust::fill(nodeInfoVecs.discretized_edges_strain.begin(), nodeInfoVecs.discretized_edges_strain.end(),0.0);
	thrust::fill(nodeInfoVecs.host_edge_right.begin(), nodeInfoVecs.host_edge_right.end(), 0);	//fill force and velocity with zeros for computation.
	thrust::fill(nodeInfoVecs.host_edge_left.begin(), nodeInfoVecs.host_edge_left.end(), 0);	//fill force and velocity with zeros for computation.
	thrust::fill(nodeInfoVecs.id_edges_made_temp.begin(), nodeInfoVecs.id_edges_made_temp.end(), 0);

	thrust::fill(nodeInfoVecs.sum_forces_on_node.begin(), nodeInfoVecs.sum_forces_on_node.end(), 0);

	thrust::fill(nodeInfoVecs.node_upper_selection_pull.begin(),
		nodeInfoVecs.node_upper_selection_pull.end(),false);

	thrust::fill(nodeInfoVecs.node_lower_selection_pull.begin(),
		nodeInfoVecs.node_lower_selection_pull.end(),false);

	thrust::copy(hostNodeInfoVecs.host_node_is_collagen.begin(), hostNodeInfoVecs.host_node_is_collagen.end(), nodeInfoVecs.node_is_collagen.begin());
	thrust::copy(hostNodeInfoVecs.host_node_is_elastin.begin(), hostNodeInfoVecs.host_node_is_elastin.end(), nodeInfoVecs.node_is_elastin.begin());
	thrust::copy(hostNodeInfoVecs.host_pos_x.begin(), hostNodeInfoVecs.host_pos_x.end(), nodeInfoVecs.node_loc_x.begin());
	thrust::copy(hostNodeInfoVecs.host_pos_y.begin(), hostNodeInfoVecs.host_pos_y.end(), nodeInfoVecs.node_loc_y.begin());
	thrust::copy(hostNodeInfoVecs.host_pos_z.begin(), hostNodeInfoVecs.host_pos_z.end(), nodeInfoVecs.node_loc_z.begin());

	nodeInfoVecs.links_made_individual_thread.resize(generalParams.max_node_count);

	nodeInfoVecs.id_temp_linked_left.resize(generalParams.max_node_count * generalParams.max_links_per_iteration);
	nodeInfoVecs.id_temp_linked_right.resize(generalParams.max_node_count * generalParams.max_links_per_iteration);
	//copy fixed positions
	nodeInfoVecs.host_id_left.resize(generalParams.max_node_count * generalParams.max_links_per_iteration);
	nodeInfoVecs.host_id_right.resize(generalParams.max_node_count * generalParams.max_links_per_iteration);

	nodeInfoVecs.is_node_fixed.resize(generalParams.max_node_count);
	thrust::fill(nodeInfoVecs.is_node_fixed.begin(), nodeInfoVecs.is_node_fixed.end(), false);

	//now that all the nodes are loaded in, choose the top to apply strain, and fix the bottom

	determine_bounds();

	//at this point all nodes are filled, so we can generate domainParams before seeding dpd particles.
	init_dim_general(
		nodeInfoVecs,
		domainParams,
		auxVecs,
		generalParams);

	//set original parameters for domain. others will be reset as simulation takes place.
	domainParams.origin_min_x = domainParams.min_x;
	domainParams.origin_max_x = domainParams.max_x;
	domainParams.origin_min_y = domainParams.min_y;
	domainParams.origin_max_y = domainParams.max_y;
	domainParams.origin_min_z = domainParams.min_z;
	domainParams.origin_max_z = domainParams.max_z;
	std::cout<< "node count : " <<nodeInfoVecs.node_loc_y.size()<< std::endl;


	auxVecs.id_bucket_net_intc.resize(generalParams.max_node_count);
	auxVecs.id_value_net_intc.resize(generalParams.max_node_count);
	auxVecs.id_value_expanded_net_intc.resize(27 * (generalParams.max_node_count));
	auxVecs.id_bucket_expanded_net_intc.resize(27 *( generalParams.max_node_count));

};

void System::determine_bounds() {
	//determin z positions of nodes to be pulled and fixed.

	thrust::device_vector<double> zPosTemp;
	zPosTemp.resize(generalParams.max_node_count);
	thrust::copy(nodeInfoVecs.node_loc_z.begin(), nodeInfoVecs.node_loc_z.end(), zPosTemp.begin());

	//not used
	//pull at least 10% of nodes.
	//unsigned tempNodeAmmount = static_cast<unsigned>( 0.25 * generalParams.max_node_count ); //pull 10% of top nodes

	//sort in increasing order
	thrust::sort(zPosTemp.begin(), zPosTemp.end(), thrust::less<double>());
	double length = zPosTemp[ zPosTemp.size()-1 ];
	std::cout<<"start end ZposTemp: "<< zPosTemp[0] << " "<< zPosTemp[zPosTemp.size()-1]<<std::endl;

	//upperLevelAlt pulls 10% default. Set in main.cpp using input
	if (generalParams.pull_percent >= 1.0) {
		std::cout<<"ERROR PULL PERCENT MUST BE LESS THAN ONE"<<std::endl;;
	}
	double upperLevelAlt = (1.0 - generalParams.pull_percent) * length;


	double lowerLevel = abs (upperLevelAlt - (zPosTemp[zPosTemp.size()-1]));

	std::cout<<"minimal level final choice for strain choice: " << lowerLevel <<std::endl;

	std::cout<<"maximal level final choice for strain choice: " << upperLevelAlt <<std::endl;

	//apply strain only to original nodes and not added edge subdivision nodes. Set top and bottom

	thrust::replace_if(nodeInfoVecs.node_upper_selection_pull.begin(), nodeInfoVecs.node_upper_selection_pull.begin() + generalParams.origin_node_count,
						nodeInfoVecs.node_loc_z.begin(),
						IsGreaterThanLevel( upperLevelAlt ), true);

	thrust::replace_if(nodeInfoVecs.node_lower_selection_pull.begin(), nodeInfoVecs.node_lower_selection_pull.begin() + generalParams.origin_node_count,
						nodeInfoVecs.node_loc_z.begin(),
						IsLessThanLevel( lowerLevel ), true);

	generalParams.numUpperStrainNodes = thrust::count_if(nodeInfoVecs.node_upper_selection_pull.begin(),nodeInfoVecs.node_upper_selection_pull.end(), IsEqualToOne( ) );
	generalParams.numLowerStrainNodes = thrust::count_if(nodeInfoVecs.node_lower_selection_pull.begin(),nodeInfoVecs.node_lower_selection_pull.end(), IsEqualToOne( ) );

	std::cout<<"number of nodes pulled for strain: " << generalParams.numLowerStrainNodes + generalParams.numUpperStrainNodes <<std::endl;

	unsigned numFixed = thrust::count_if(nodeInfoVecs.is_node_fixed.begin(),nodeInfoVecs.is_node_fixed.end(), IsEqualToOne() );
	std::cout<<"number of nodes fixed: " << numFixed <<std::endl;
	zPosTemp.resize(0);

}

void System::set_bend_vecs(
	HostNodeInfoVecs& hostNodeInfoVecs) {

	bendInfoVecs.leftIndex.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.centerIndex.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.rightIndex.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.angleZero.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);

	thrust::fill(bendInfoVecs.leftIndex.begin(),bendInfoVecs.leftIndex.end(),ULONG_MAX);
	thrust::fill(bendInfoVecs.centerIndex.begin(),bendInfoVecs.centerIndex.end(),ULONG_MAX);
	thrust::fill(bendInfoVecs.rightIndex.begin(),bendInfoVecs.rightIndex.end(),ULONG_MAX);

	//after default value is set, set the real id's
	thrust::copy(hostNodeInfoVecs.host_torsion_index_left.begin(), hostNodeInfoVecs.host_torsion_index_left.end(), bendInfoVecs.leftIndex.begin());
	thrust::copy(hostNodeInfoVecs.host_torsion_index_center.begin(), hostNodeInfoVecs.host_torsion_index_center.end(), bendInfoVecs.centerIndex.begin());
	thrust::copy(hostNodeInfoVecs.host_torsion_index_right.begin(), hostNodeInfoVecs.host_torsion_index_right.end(), bendInfoVecs.rightIndex.begin());

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				bendInfoVecs.leftIndex.begin(),
				bendInfoVecs.centerIndex.begin(),
				bendInfoVecs.rightIndex.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				bendInfoVecs.leftIndex.begin(),
				bendInfoVecs.centerIndex.begin(),
				bendInfoVecs.rightIndex.begin())) + bendInfoVecs.total_bend_count,
			bendInfoVecs.angleZero.begin(),//save vector
		functor_initial_angle(
			thrust::raw_pointer_cast(nodeInfoVecs.node_loc_x.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_loc_y.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_loc_z.data())));

	//		std::cout<<" in NSD device values"<<std::endl;
	for (unsigned i = 0; i<bendInfoVecs.total_bend_count; i++) {
		unsigned n0 = bendInfoVecs.leftIndex[i];
		unsigned n1 = bendInfoVecs.centerIndex[i];
		unsigned n2 = bendInfoVecs.rightIndex[i];
		std::cout<< "angle : "<< n0<< " " << n1<< " " << n2<< " " << bendInfoVecs.angleZero[i]<<std::endl;
	}

	//3x bigger since each spring affects 3 nodes.
	bendInfoVecs.forceX.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.forceY.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.forceZ.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.tempForceX.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.tempForceY.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.tempForceZ.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);


	thrust::fill(bendInfoVecs.forceX.begin(), bendInfoVecs.forceX.end(), 0.0);
	thrust::fill(bendInfoVecs.forceY.begin(), bendInfoVecs.forceY.end(), 0.0);
	thrust::fill(bendInfoVecs.forceZ.begin(), bendInfoVecs.forceZ.end(), 0.0);

	bendInfoVecs.tempTorIndices.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.reducedIds.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);


};

void System::set_edge_vecs(
	HostNodeInfoVecs& hostNodeInfoVecs ) {

	edgeInfoVecs.global_neighbors.resize(generalParams.max_node_count * generalParams.max_nbr_count);
  	edgeInfoVecs.global_isedge_collagen.resize(generalParams.max_node_count * generalParams.max_nbr_count);
  	edgeInfoVecs.global_isedge_elastin.resize(generalParams.max_node_count * generalParams.max_nbr_count);

	edgeInfoVecs.current_node_edge_count_vec.resize(generalParams.max_node_count);

	edgeInfoVecs.global_length_zero.resize(generalParams.max_node_count * generalParams.max_nbr_count);
	edgeInfoVecs.num_origin_nbr_per_node_vec.resize(generalParams.max_node_count);


  	thrust::fill(edgeInfoVecs.global_neighbors.begin(), edgeInfoVecs.global_neighbors.end(), generalParams.max_node_count);
  	thrust::fill(edgeInfoVecs.global_isedge_collagen.begin(), edgeInfoVecs.global_isedge_collagen.end(), false);
  	thrust::fill(edgeInfoVecs.global_isedge_elastin.begin(), edgeInfoVecs.global_isedge_elastin.end(), false);

  	thrust::fill(edgeInfoVecs.current_node_edge_count_vec.begin(), edgeInfoVecs.current_node_edge_count_vec.end(),0);
	thrust::fill(edgeInfoVecs.global_length_zero.begin(), edgeInfoVecs.global_length_zero.end(), 0.0);



	nodeInfoVecs.host_edge_left = hostNodeInfoVecs.host_spring_edge_left;
	nodeInfoVecs.host_edge_right = hostNodeInfoVecs.host_spring_edge_right;
	//scan through hostAdj and put in device.
	for (unsigned id = 0; id < hostNodeInfoVecs.host_spring_length_zero.size(); id++) {
		generalParams.totalNumberOfEdges++;
		unsigned idL = hostNodeInfoVecs.host_spring_edge_left[id];
		unsigned idR = hostNodeInfoVecs.host_spring_edge_right[id];

     	bool is_idL_collagen = hostNodeInfoVecs.host_node_is_collagen[idL];
		bool is_idR_collagen = hostNodeInfoVecs.host_node_is_collagen[idR];
		bool is_edge_collagen = false;
		bool is_edge_elastin = false;
		if (is_idL_collagen && is_idR_collagen) {
			is_edge_collagen=true;
		}
		else { is_edge_elastin = true;}
		//std::cout<< "linking " << idL << " to " <<idR << std::endl;

		 double edgeLen = hostNodeInfoVecs.host_spring_length_zero[id];
				//we use the global_length_zero vector to identify edges as well.

    	//node id is row, column node is connected to row node.
		//add edge for left node
		unsigned edgeNumL = edgeInfoVecs.current_node_edge_count_vec[idL]; //number of edges on (nodeId = row)	is that entry in cECV
		unsigned indexL = idL*generalParams.max_nbr_count + edgeNumL;
		edgeInfoVecs.global_length_zero[indexL] = edgeLen;
		edgeInfoVecs.global_neighbors[indexL] = idR;
		edgeInfoVecs.global_isedge_collagen[indexL] = is_edge_collagen;
		edgeInfoVecs.global_isedge_elastin[indexL] = is_edge_elastin;

		(edgeInfoVecs.current_node_edge_count_vec[idL])++; //right connects to left

		//add edge for right node
		unsigned edgeNumR = edgeInfoVecs.current_node_edge_count_vec[idR]; //number of edges on (nodeId = row)	is that entry in cECV
		unsigned indexR = idR*generalParams.max_nbr_count + edgeNumR;
		edgeInfoVecs.global_length_zero[indexR] = edgeLen;
		edgeInfoVecs.global_neighbors[indexR] = idL;
		edgeInfoVecs.global_isedge_collagen[indexR] = is_edge_collagen;
		edgeInfoVecs.global_isedge_elastin[indexR] = is_edge_elastin;

		(edgeInfoVecs.current_node_edge_count_vec[idR])++; //left connects to right
		generalParams.current_edge_count += 1;
	}
	//at this point current_node_edge_count_vec holds the number of edges, copy this to
	thrust::copy(edgeInfoVecs.current_node_edge_count_vec.begin(), edgeInfoVecs.current_node_edge_count_vec.end(), edgeInfoVecs.num_origin_nbr_per_node_vec.begin());
};

void System::set_extras() {
	extensionParams.originalNetworkLength = domainParams.max_z; //compression along x extensionParams.axis
	extensionParams.originalNetworkWidth = domainParams.max_x;  //strain along z extensionParams.axis.
};
