#include "hip/hip_runtime.h"


#include "system.h"
#include "system_builder.h"

#include "system_structures.h"
#include "storage.h"
#include <numeric>

Storage::Storage(std::weak_ptr<System> a_system,
	std::weak_ptr<System_Builder> b_system , const std::string& a_fileName) {
	
	system = a_system;
	builder = b_system;
	bn = a_fileName; //this will be used later to open files
	std::ofstream statesOutput(a_fileName + ".sta");
	std::ofstream statesOutputStrain(a_fileName + "_Strain.sta");

	std::shared_ptr<System> sysA = system.lock();
	std::shared_ptr<System_Builder> sysB = builder.lock();

	if ((sysA) && (sysB) ){
		unsigned max_node_count = sysA->generalParams.max_node_count;
		unsigned max_nbr_count = sysA->generalParams.max_nbr_count;

		statesOutput << "node_count " << max_node_count << '\n';
		statesOutput << "origin_node_count " << sysA->generalParams.origin_node_count << '\n';
		statesOutput << "origin_link_count " << sysA->generalParams.origin_edge_count << '\n';
		statesOutput << "sub_node_count " << sysA->generalParams.sub_node_count << std::endl;//system->getSubNodesSize() << '\n';
		statesOutput << "link_count " << sysA->generalParams.current_edge_count << '\n';

		for (unsigned edge = 0; edge < sysB->hostNodeInfoVecs.host_spring_edge_left.size(); edge++) {
			unsigned idLeft = sysB->hostNodeInfoVecs.host_spring_edge_left[edge];
			unsigned idRight = sysB->hostNodeInfoVecs.host_spring_edge_right[edge];
			statesOutput << '\n' << idLeft << ' ' << idRight;
		}

	}


	statesOutput.close();
}

void Storage::updateStrain() {

/*	std::shared_ptr<system> sys = system.lock();
	if (sys) {

	statesOutputStrain.open(bn + "_Strain.sta", std::ofstream::out | std::ofstream::app);
		statesOutputStrain << "\ntime " << sys->generalParams.currentTime;
		statesOutputStrain << "\nforce " << sys->extensionParams.totalAppliedForce;

		statesOutputStrain << "\nupper_XPos " << sys->domainParams.max_x;
		statesOutputStrain << "\nlower_XPos " << sys->domainParams.min_x;

		statesOutputStrain << "\nupper_YPos " << sys->domainParams.max_y;
		statesOutputStrain << "\nlower_YPos " << sys->domainParams.min_y;

		statesOutputStrain << "\nupper_ZPosAve " << sys->extensionParams.averageUpperStrain;
		statesOutputStrain << "\nlower_ZPosAve " << sys->extensionParams.averageLowerStrain;
		statesOutputStrain << "\noriginal_extended_percent " << sys->edgeInfoVecs.percentOriginalEdgesExtended;
		statesOutputStrain << "\noriginal_compressed_percent " << sys->edgeInfoVecs.percentOriginalEdgesCompressed;
		statesOutputStrain << "\noriginal_average_strain " << sys->edgeInfoVecs.averageStrainOriginalEdges;


		for (unsigned i = 0; i < sys->edgeInfoVecs.strainBucketOriginalNeg.size(); i++ ) {
			statesOutputStrain << " \noriginal_strain_neg " << sys->edgeInfoVecs.strainBucketOriginalNeg[i] / (2.0 * sys->generalParams.originEdgeCount);

		}
		for (unsigned i = 0; i < sys->edgeInfoVecs.strainBucketOriginalPos.size(); i++ ) {
			statesOutputStrain << " \noriginal_strain_pos " << sys->edgeInfoVecs.strainBucketOriginalPos[i] /  (2.0 * sys->generalParams.originEdgeCount);
		}




		statesOutputStrain << "\nadded_extended_percent " << sys->edgeInfoVecs.percentAddedEdgesExtended;
		statesOutputStrain << "\nadded_compressed_percent " << sys->edgeInfoVecs.percentAddedEdgesCompressed;
		statesOutputStrain << "\nadded_average_strain " << sys->edgeInfoVecs.averageStrainAddedEdges;

		double sumOfNumsAdded = std::accumulate(sys->edgeInfoVecs.strainBucketAddedNeg.begin(),
			sys->edgeInfoVecs.strainBucketAddedNeg.end(),0.0);
		sumOfNumsAdded += std::accumulate(sys->edgeInfoVecs.strainBucketAddedPos.begin(),
			sys->edgeInfoVecs.strainBucketAddedPos.end(),0.0);
		for (unsigned i = 0; i < sys->edgeInfoVecs.strainBucketAddedNeg.size(); i++ ) {
			statesOutputStrain << " \nadded_strain_neg " << sys->edgeInfoVecs.strainBucketAddedNeg[i]/sumOfNumsAdded;

		}
		for (unsigned i = 0; i < sys->edgeInfoVecs.strainBucketAddedPos.size(); i++ ) {
			statesOutputStrain << " \nadded_strain_pos " << sys->edgeInfoVecs.strainBucketAddedPos[i]/sumOfNumsAdded;

		}


		for (unsigned i = 0; i < sys->edgeInfoVecs.alignmentAverage.size(); i++ ) {
			double numEdgesInBin = sys->edgeInfoVecs.numberOfEdgesAlignment[i];
			double val = 0.0;

			if (numEdgesInBin != 0.0) {
				val  = sys->edgeInfoVecs.alignmentAverage[i]/numEdgesInBin;
			}
			statesOutputStrain << " \nslice_alignment " << val;

		}


	}
	statesOutputStrain.flush();
	statesOutputStrain.close();*/

};

void Storage::updateTotalStrain(void) {
	std::shared_ptr<System> sys = system.lock();
	if (sys) {

		double currentStrain = (sys->extensionParams.averageUpperStrain - sys->extensionParams.averageLowerStrain) /
			 (sys->extensionParams.originAverageUpperStrain - sys->extensionParams.originAverageLowerStrain ) - 1.0;
		//first create a new file using the current network strain

		std::string format = ".sta";
		std::string strain =  std::to_string(currentStrain);
		std::string initial = "StrainTest/Strain_";
		std::ofstream ofs;
		std::string Filename = initial + strain + format;
		ofs.open(Filename.c_str());



		unsigned max_nbr_count = sys->generalParams.max_nbr_count;
		unsigned max_node_count = sys->generalParams.max_node_count;
		unsigned originalNodeCount = sys->generalParams.origin_node_count;
		unsigned originalEdgeCount = sys->generalParams.origin_edge_count;
		unsigned edgeCountDiscretize = sys->generalParams.current_edge_count;
		//Now first place strain
		ofs << std::setprecision(5) <<std::fixed<< "time " << sys->generalParams.currentTime<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "network_strain " << currentStrain<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "min_x " << sys->domainParams.min_x<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "max_x " << sys->domainParams.max_x<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "min_y " << sys->domainParams.min_y<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "max_y " << sys->domainParams.max_y<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "min_z " << sys->domainParams.min_x<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "max_z " << sys->domainParams.max_x<<std::endl;


		//ofs << std::setprecision(5) <<std::fixed<< "total_applied_force " << sys->extensionParams.totalAppliedForce<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "original_node_count " << originalNodeCount <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "node_count_discretize " << max_node_count <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "original_edge_count " << originalEdgeCount <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "edge_count_discretize " << edgeCountDiscretize <<std::endl;

		//place nodes
		for (unsigned i = 0; i < sys->nodeInfoVecs.node_loc_x.size(); i++) {
			double x = sys->nodeInfoVecs.node_loc_x[i];
			double y = sys->nodeInfoVecs.node_loc_y[i];
			double z = sys->nodeInfoVecs.node_loc_z[i];
			ofs << std::setprecision(5) <<std::fixed<< "node " << x << " " << y << " " << z <<std::endl;

		}
		//place force node is experiencing
		for (unsigned i = 0; i < sys->nodeInfoVecs.node_loc_x.size(); i++) {
			ofs << std::setprecision(5) <<std::fixed<< "force_on_node " << sys->nodeInfoVecs.sum_forces_on_node[i]<<std::endl;

		}

		//place original edges
		for (unsigned edge = 0; edge < sys->generalParams.origin_edge_count; edge++) {
			unsigned idL = sys->nodeInfoVecs.host_edge_left[edge];
			unsigned idR = sys->nodeInfoVecs.host_edge_right[edge];
			ofs <<"original_edge_discretized " <<idL <<" "<< idR <<std::endl;

		}

		//place added edges
		for (unsigned edge = sys->generalParams.origin_edge_count; edge < sys->generalParams.current_edge_count; edge++) {
			unsigned idL = sys->nodeInfoVecs.host_edge_left[edge];
			unsigned idR = sys->nodeInfoVecs.host_edge_right[edge];
			ofs <<"added_edge " <<idL <<" "<< idR <<std::endl;

		}

		//original edge strain
		for (unsigned i = 0; i < sys->generalParams.origin_edge_count; i++ ){
			double val = sys->nodeInfoVecs.discretized_edges_strain[i];

			ofs << std::setprecision(5)<< std::fixed<<"original_edge_strain " << val <<std::endl;
		}

		//original edge alignment
		for (unsigned i = 0; i < sys->generalParams.origin_edge_count; i++ ){
			double val = sys->nodeInfoVecs.discretized_edges_alignment[i];
			ofs << std::setprecision(5)<< std::fixed<<"original_edge_alignment " << val <<std::endl;
		}

		//added edge strain
		for (unsigned i = sys->generalParams.origin_edge_count; i < sys->generalParams.current_edge_count; i++ ){
			double val = sys->nodeInfoVecs.discretized_edges_strain[i];
			ofs << std::setprecision(5)<< std::fixed<<"added_edge_strain " << val <<std::endl;
		}

		//added links per node.
		for (unsigned i = 0; i < sys->generalParams.max_node_count; i++ ){
			unsigned val = sys->edgeInfoVecs.current_node_edge_count_vec[i] -
				sys->edgeInfoVecs.num_origin_nbr_per_node_vec[i];
			ofs << std::setprecision(5)<< std::fixed<<"bind_sites_per_node " << val <<std::endl;
		}



	}
}


void Storage::print_VTK_File() {

	std::shared_ptr<System> sys = system.lock();
	if (sys) {
		iteration+=1;
		unsigned digits = ceil(log10(iteration + 1));
		std::string format = ".vtk";
		std::string Number;
		std::string initial = "AnimationTest/Network_";
		std::ofstream ofs;
		if (digits == 1 || digits == 0) {
			Number = "0000" + std::to_string(iteration);
		}
		else if (digits == 2) {
			Number = "000" + std::to_string(iteration);
		}
		else if (digits == 3) {
			Number = "00" + std::to_string(iteration);
		}
		else if (digits == 4) {
			Number = "0" + std::to_string(iteration);
		}

		std::string Filename = initial + Number + format;

		ofs.open(Filename.c_str());


		unsigned max_node_count = sys->generalParams.max_node_count;
		unsigned max_nbr_count = sys->generalParams.max_nbr_count;
		unsigned numEdges = sys->generalParams.current_edge_count;

		ofs << "# vtk DataFile Version 3.0" << std::endl;
		ofs << "Point representing Sub_cellular elem model" << std::endl;
		ofs << "ASCII" << std::endl << std::endl;
		ofs << "DATASET UNSTRUCTURED_GRID" << std::endl;


		ofs << "POINTS " << max_node_count << " float" << std::endl;
		for (unsigned i = 0; i< max_node_count; i++) {
			double xPos = sys->nodeInfoVecs.node_loc_x[i];
			double yPos = sys->nodeInfoVecs.node_loc_y[i];
			double zPos = sys->nodeInfoVecs.node_loc_z[i];

			ofs << std::setprecision(5) <<std::fixed<< xPos << " " << yPos << " " << zPos << " " << '\n'<< std::fixed;
		}
		//now plot particles
		unsigned numCells = numEdges;
		unsigned numNumsInCells = 3 * numEdges;


		ofs << "CELLS " << numCells << " " << numNumsInCells << std::endl;

		for (unsigned idA = 0; idA < max_node_count; idA++ ){

			unsigned beginIndex = idA * max_nbr_count;
			unsigned endIndex = beginIndex + max_nbr_count;
			for (unsigned i = beginIndex; i < endIndex; i++) {//currentSpringCount is the length of index and value vectors
				unsigned idB = sys->edgeInfoVecs.global_neighbors[i];//look through possible neighbors. May contain ULONG_MAX

				if ((idA < idB) && (idB < max_node_count) ) {
					ofs<< 2 << " " << idA << " " << idB << std::endl;
				}
			}
		}

		ofs << "CELL_TYPES " << numCells << std::endl;
		for (unsigned i = 0; i<numEdges; i++) {
			ofs << 3 << std::endl;
		}

		//
		ofs << "CELL_DATA " << numCells << std::endl;
		ofs << "SCALARS Fiber_Strain double " << std::endl;
		ofs << "LOOKUP_TABLE default "  << std::endl;
		for (unsigned edge = 0; edge < numEdges; edge++) {
			unsigned idA = sys->nodeInfoVecs.host_edge_left[edge];
			unsigned idB = sys->nodeInfoVecs.host_edge_right[edge];

			unsigned begin = idA * sys->generalParams.max_nbr_count;
			unsigned end = begin + sys->generalParams.max_nbr_count;
			double L0;
			for (unsigned i = begin; i < end; i++) {
				unsigned idTemp = sys->edgeInfoVecs.global_neighbors[i];
				if (idTemp == idB){
					L0 = sys->edgeInfoVecs.global_length_zero[i];
				}
			}
			double xL = sys->nodeInfoVecs.node_loc_x[idA];
			double yL = sys->nodeInfoVecs.node_loc_y[idA];
			double zL = sys->nodeInfoVecs.node_loc_z[idA];
			double xR = sys->nodeInfoVecs.node_loc_x[idB];
			double yR = sys->nodeInfoVecs.node_loc_y[idB];
			double zR = sys->nodeInfoVecs.node_loc_z[idB];

			double L1 = std::sqrt( (xL - xR)*(xL - xR)+(yL - yR)*(yL - yR)+(zL - zR)*(zL - zR));
			double strain = (L1 - L0) / L0;
			ofs << std::fixed << strain   << std::endl;

		}

		ofs.close();

	}
};
