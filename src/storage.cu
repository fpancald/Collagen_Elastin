#include "hip/hip_runtime.h"


#include "system.h"
#include "system_builder.h"

#include "system_structures.h"
#include "storage.h"
#include <numeric>

Storage::Storage(std::weak_ptr<system> a_system,
	std::weak_ptr<system_builder> b_system , const std::string& a_fileName) {
	//std::cout << "FDM constructor" << std::endl;
	system = a_system;
	builder = b_system;
	bn = a_fileName; //this will be used later to open files
	std::ofstream statesOutput(a_fileName + ".sta");
	std::ofstream statesOutputStrain(a_fileName + "_Strain.sta");

	std::shared_ptr<system> sysA = system.lock();
	std::shared_ptr<system_builder> sysB = builder.lock();

	if ((sysA) && (sysB) ){
		unsigned max_node_count = sysA->generalParams.max_node_count;
		unsigned max_nbr_count = sysA->generalParams.max_nbr_count;

		statesOutput << "node_count " << max_node_count << '\n';
		statesOutput << "origin_node_count " << sysA->generalParams.originNodeCount << '\n';
		statesOutput << "origin_link_count " << sysA->generalParams.originLinkCount << '\n';
		statesOutput << "sub_node_count " << sysA->generalParams.sub_node_count << std::endl;//system->getSubNodesSize() << '\n';
		statesOutput << "link_count " << sysA->generalParams.originEdgeCount << '\n';

		for (unsigned edge = 0; edge < sysB->hostNodeInfoVecs.host_spring_edge_left.size(); edge++) {
			unsigned idLeft = sysB->hostNodeInfoVecs.host_spring_edge_left[edge];
			unsigned idRight = sysB->hostNodeInfoVecs.host_spring_edge_right[edge];
			statesOutput << '\n' << idLeft << ' ' << idRight;
		}

	}


	statesOutput.close();
}

void Storage::updateStrain() {

/*	std::shared_ptr<system> sys = system.lock();
	if (sys) {

	statesOutputStrain.open(bn + "_Strain.sta", std::ofstream::out | std::ofstream::app);
		statesOutputStrain << "\ntime " << sys->generalParams.currentTime;
		statesOutputStrain << "\nforce " << sys->extensionParams.totalAppliedForce;

		statesOutputStrain << "\nupper_XPos " << sys->domainParams.max_x;
		statesOutputStrain << "\nlower_XPos " << sys->domainParams.min_x;

		statesOutputStrain << "\nupper_YPos " << sys->domainParams.max_y;
		statesOutputStrain << "\nlower_YPos " << sys->domainParams.min_y;

		statesOutputStrain << "\nupper_ZPosAve " << sys->extensionParams.averageUpperStrain;
		statesOutputStrain << "\nlower_ZPosAve " << sys->extensionParams.averageLowerStrain;
		statesOutputStrain << "\noriginal_extended_percent " << sys->edgeInfoVecs.percentOriginalEdgesExtended;
		statesOutputStrain << "\noriginal_compressed_percent " << sys->edgeInfoVecs.percentOriginalEdgesCompressed;
		statesOutputStrain << "\noriginal_average_strain " << sys->edgeInfoVecs.averageStrainOriginalEdges;


		for (unsigned i = 0; i < sys->edgeInfoVecs.strainBucketOriginalNeg.size(); i++ ) {
			statesOutputStrain << " \noriginal_strain_neg " << sys->edgeInfoVecs.strainBucketOriginalNeg[i] / (2.0 * sys->generalParams.originEdgeCount);

		}
		for (unsigned i = 0; i < sys->edgeInfoVecs.strainBucketOriginalPos.size(); i++ ) {
			statesOutputStrain << " \noriginal_strain_pos " << sys->edgeInfoVecs.strainBucketOriginalPos[i] /  (2.0 * sys->generalParams.originEdgeCount);
		}




		statesOutputStrain << "\nadded_extended_percent " << sys->edgeInfoVecs.percentAddedEdgesExtended;
		statesOutputStrain << "\nadded_compressed_percent " << sys->edgeInfoVecs.percentAddedEdgesCompressed;
		statesOutputStrain << "\nadded_average_strain " << sys->edgeInfoVecs.averageStrainAddedEdges;

		double sumOfNumsAdded = std::accumulate(sys->edgeInfoVecs.strainBucketAddedNeg.begin(),
			sys->edgeInfoVecs.strainBucketAddedNeg.end(),0.0);
		sumOfNumsAdded += std::accumulate(sys->edgeInfoVecs.strainBucketAddedPos.begin(),
			sys->edgeInfoVecs.strainBucketAddedPos.end(),0.0);
		for (unsigned i = 0; i < sys->edgeInfoVecs.strainBucketAddedNeg.size(); i++ ) {
			statesOutputStrain << " \nadded_strain_neg " << sys->edgeInfoVecs.strainBucketAddedNeg[i]/sumOfNumsAdded;

		}
		for (unsigned i = 0; i < sys->edgeInfoVecs.strainBucketAddedPos.size(); i++ ) {
			statesOutputStrain << " \nadded_strain_pos " << sys->edgeInfoVecs.strainBucketAddedPos[i]/sumOfNumsAdded;

		}


		for (unsigned i = 0; i < sys->edgeInfoVecs.alignmentAverage.size(); i++ ) {
			double numEdgesInBin = sys->edgeInfoVecs.numberOfEdgesAlignment[i];
			double val = 0.0;

			if (numEdgesInBin != 0.0) {
				val  = sys->edgeInfoVecs.alignmentAverage[i]/numEdgesInBin;
			}
			statesOutputStrain << " \nslice_alignment " << val;

		}


	}
	statesOutputStrain.flush();
	statesOutputStrain.close();*/

};

void Storage::updateTotalStrain(void) {
	std::shared_ptr<system> sys = system.lock();
	if (sys) {

		double currentStrain = (sys->extensionParams.averageUpperStrain - sys->extensionParams.averageLowerStrain) /
			 (sys->extensionParams.originAverageUpperStrain - sys->extensionParams.originAverageLowerStrain ) - 1.0;
		//first create a new file using the current network strain

		std::string format = ".sta";
		std::string strain =  std::to_string(currentStrain);
		std::string initial = "StrainTest/Strain_";
		std::ofstream ofs;
		std::string Filename = initial + strain + format;
		ofs.open(Filename.c_str());



		unsigned max_nbr_count = sys->generalParams.max_nbr_count;
		unsigned max_node_count = sys->generalParams.max_node_count;
		unsigned originalNodeCount = sys->generalParams.originNodeCount;
		unsigned originalEdgeCount = sys->generalParams.originLinkCount;
		unsigned edgeCountDiscretize = sys->generalParams.originEdgeCount;
		//Now first place strain
		ofs << std::setprecision(5) <<std::fixed<< "time " << sys->generalParams.currentTime<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "network_strain " << currentStrain<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "min_x " << sys->domainParams.min_x<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "max_x " << sys->domainParams.max_x<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "min_y " << sys->domainParams.min_y<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "max_y " << sys->domainParams.max_y<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "min_z " << sys->domainParams.min_x<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "max_z " << sys->domainParams.max_x<<std::endl;


		ofs << std::setprecision(5) <<std::fixed<< "total_applied_force " << sys->extensionParams.totalAppliedForce<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "original_node_count " << originalNodeCount <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "node_count_discretize " << max_node_count <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "original_edge_count " << originalEdgeCount <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "edge_count_discretize " << edgeCountDiscretize <<std::endl;

		//place nodes
		for (unsigned i = 0; i < sys->nodeInfoVecs.node_loc_x.size(); i++) {
			double x = sys->nodeInfoVecs.node_loc_x[i];
			double y = sys->nodeInfoVecs.node_loc_y[i];
			double z = sys->nodeInfoVecs.node_loc_z[i];
			ofs << std::setprecision(5) <<std::fixed<< "node " << x << " " << y << " " << z <<std::endl;

		}
		//place force node is experiencing
		for (unsigned i = 0; i < sys->nodeInfoVecs.node_loc_x.size(); i++) {
			ofs << std::setprecision(5) <<std::fixed<< "force_on_node " << sys->nodeInfoVecs.sum_forces_on_node[i]<<std::endl;

		}

		//place original edges
		for (unsigned edge = 0; edge < sys->generalParams.originEdgeCount; edge++) {
			unsigned idL = sys->nodeInfoVecs.device_edge_left[edge];
			unsigned idR = sys->nodeInfoVecs.device_edge_right[edge];
			ofs <<"original_edge_discretized " <<idL <<" "<< idR <<std::endl;

		}

		//place added edges
		for (unsigned edge = sys->generalParams.originEdgeCount; edge < sys->generalParams.currentEdgeCount; edge++) {
			unsigned idL = sys->nodeInfoVecs.device_edge_left[edge];
			unsigned idR = sys->nodeInfoVecs.device_edge_right[edge];
			ofs <<"added_edge " <<idL <<" "<< idR <<std::endl;

		}

		//original edge strain
		for (unsigned i = 0; i < sys->generalParams.originEdgeCount; i++ ){
			double val = sys->nodeInfoVecs.discretized_edges_strain[i];

			ofs << std::setprecision(5)<< std::fixed<<"original_edge_strain " << val <<std::endl;
		}

		//original edge alignment
		for (unsigned i = 0; i < sys->generalParams.originEdgeCount; i++ ){
			double val = sys->nodeInfoVecs.discretized_edges_alignment[i];
			ofs << std::setprecision(5)<< std::fixed<<"original_edge_alignment " << val <<std::endl;
		}

		//added edge strain
		for (unsigned i = sys->generalParams.originEdgeCount; i < sys->generalParams.currentEdgeCount; i++ ){
			double val = sys->nodeInfoVecs.discretized_edges_strain[i];
			ofs << std::setprecision(5)<< std::fixed<<"added_edge_strain " << val <<std::endl;
		}

		//added links per node.
		for (unsigned i = 0; i < sys->generalParams.max_node_count; i++ ){
			unsigned val = sys->edgeInfoVecs.current_node_edge_count_vec[i] -
				sys->edgeInfoVecs.num_origin_nbr_per_node_vec[i];
			ofs << std::setprecision(5)<< std::fixed<<"bind_sites_per_node " << val <<std::endl;
		}



	}
}


void Storage::print_VTK_File() {

	std::shared_ptr<system> sys = system.lock();
	if (sys) {
		iteration+=1;
		unsigned digits = ceil(log10(iteration + 1));
		std::string format = ".vtk";
		std::string Number;
		std::string initial = "AnimationTest/Network_";
		std::ofstream ofs;
		if (digits == 1 || digits == 0) {
			Number = "0000" + std::to_string(iteration);
		}
		else if (digits == 2) {
			Number = "000" + std::to_string(iteration);
		}
		else if (digits == 3) {
			Number = "00" + std::to_string(iteration);
		}
		else if (digits == 4) {
			Number = "0" + std::to_string(iteration);
		}

		std::string Filename = initial + Number + format;

		ofs.open(Filename.c_str());


		unsigned max_node_count = sys->generalParams.max_node_count;
		unsigned max_nbr_count = sys->generalParams.max_nbr_count;
		unsigned numEdges = sys->generalParams.currentEdgeCount;

		ofs << "# vtk DataFile Version 3.0" << std::endl;
		ofs << "Point representing Sub_cellular elem model" << std::endl;
		ofs << "ASCII" << std::endl << std::endl;
		ofs << "DATASET UNSTRUCTURED_GRID" << std::endl;


		ofs << "POINTS " << max_node_count << " float" << std::endl;
		for (unsigned i = 0; i< max_node_count; i++) {
			double xPos = sys->nodeInfoVecs.node_loc_x[i];
			double yPos = sys->nodeInfoVecs.node_loc_y[i];
			double zPos = sys->nodeInfoVecs.node_loc_z[i];

			ofs << std::setprecision(5) <<std::fixed<< xPos << " " << yPos << " " << zPos << " " << '\n'<< std::fixed;
		}
		//now plot particles
		unsigned numCells = numEdges;
		unsigned numNumsInCells = 3 * numEdges;


		ofs << "CELLS " << numCells << " " << numNumsInCells << std::endl;

		for (unsigned idA = 0; idA < max_node_count; idA++ ){

			unsigned beginIndex = idA * max_nbr_count;
			unsigned endIndex = beginIndex + max_nbr_count;
			for (unsigned i = beginIndex; i < endIndex; i++) {//currentSpringCount is the length of index and value vectors
				unsigned idB = sys->edgeInfoVecs.global_neighbors[i];//look through possible neighbors. May contain ULONG_MAX

				if ((idA < idB) && (idB < max_node_count) ) {
					ofs<< 2 << " " << idA << " " << idB << std::endl;
				}
			}
		}

		ofs << "CELL_TYPES " << numCells << std::endl;
		for (unsigned i = 0; i<numEdges; i++) {
			ofs << 3 << std::endl;
		}

		//
		ofs << "CELL_DATA " << numCells << std::endl;
		ofs << "SCALARS Fiber_Strain double " << std::endl;
		ofs << "LOOKUP_TABLE default "  << std::endl;
		for (unsigned edge = 0; edge < numEdges; edge++) {
			unsigned idA = sys->nodeInfoVecs.device_edge_left[edge];
			unsigned idB = sys->nodeInfoVecs.device_edge_right[edge];

			unsigned begin = idA * sys->generalParams.max_nbr_count;
			unsigned end = begin + sys->generalParams.max_nbr_count;
			double L0;
			for (unsigned i = begin; i < end; i++) {
				unsigned idTemp = sys->edgeInfoVecs.global_neighbors[i];
				if (idTemp == idB){
					L0 = sys->edgeInfoVecs.global_length_zero[i];
				}
			}
			double xL = sys->nodeInfoVecs.node_loc_x[idA];
			double yL = sys->nodeInfoVecs.node_loc_y[idA];
			double zL = sys->nodeInfoVecs.node_loc_z[idA];
			double xR = sys->nodeInfoVecs.node_loc_x[idB];
			double yR = sys->nodeInfoVecs.node_loc_y[idB];
			double zR = sys->nodeInfoVecs.node_loc_z[idB];

			double L1 = std::sqrt( (xL - xR)*(xL - xR)+(yL - yR)*(yL - yR)+(zL - zR)*(zL - zR));
			double strain = (L1 - L0) / L0;
			ofs << std::fixed << strain   << std::endl;

		}

		ofs.close();

	}
}

void Storage::updateStorage() {

	std::shared_ptr<system> sys = system.lock();
	if (sys) {
		statesOutput.open(bn + ".sta", std::ofstream::out | std::ofstream::app);
		//output.open(bn + ".grm", std::ofstream::out | std::ofstream::app);
		statesOutput << "\nextended percent " << sys->edgeInfoVecs.percentOriginalEdgesExtended;
		statesOutput << "\nforce " << sys->extensionParams.totalAppliedForce;
		statesOutput << "\ntime " << sys->generalParams.currentTime;
		statesOutput << "\nadded edges " << ((sys->nodeInfoVecs.id_edges_made_host).size());

		unsigned max_node_count = sys->generalParams.max_node_count;

		//print new added edges	for current time step recording

		for (unsigned i = 0; i < (sys->nodeInfoVecs.id_edges_made_host.size()); i++) {
			unsigned idUpper = sys->nodeInfoVecs.id_edges_made_host[i];
			if (idUpper != 0) {
				unsigned first = idUpper - max_node_count*(idUpper / max_node_count); //represents column
				unsigned second = (idUpper / max_node_count); //represents row

					statesOutput << '\n' << first << ' ' << second;
			}

		}




		for (unsigned i = 0; i < max_node_count; ++i) {


			double xPos = sys->nodeInfoVecs.node_loc_x[i];
			double yPos = sys->nodeInfoVecs.node_loc_y[i];
			double zPos = sys->nodeInfoVecs.node_loc_z[i];
			double xForce = sys->nodeInfoVecs.node_vel_x[i];
			double yForce = sys->nodeInfoVecs.node_vel_y[i];
			double zForce = sys->nodeInfoVecs.node_vel_z[i];
			double sumOfForces = sys->nodeInfoVecs.sum_forces_on_node[i];
			statesOutput << '\n' << i;

			//auto pos = node->getPosition();
			//auto vel = node->getVelocity();

			//for (int k = 0; k < 3; ++k)
			statesOutput << ' ' << xPos << ' ' << yPos << ' ' << zPos;

			//for (int k = 0; k < 3; ++k)
			statesOutput << ' ' << xForce << ' ' << yForce << ' ' << zForce << ' ' << sumOfForces;

		}
	}

	output << magnitudeForce << ' ' << std::endl;
	statesOutput.flush();
	output.flush();
	statesOutput.close();
	output.close();

	std::cout << "*** one step completed ***\n\n";

}
