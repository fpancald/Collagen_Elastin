#include "system_structures.h"
#include "system.h"
#include "external_force.h"
#include "functor_strain.h"
#include "functor_external_force.h"


void external_force(
	NodeInfoVecs& nodeInfoVecs,
	GeneralParams& generalParams,
	ExtensionParams& extensionParams,
	DomainParams& domainParams){
/*
	for (unsigned i = 0; i < nodeInfoVecs.node_is_collagen.size(); i++) {
		std::cout<< "is collagen: " << nodeInfoVecs.node_is_collagen[i] << std::endl;
		std::cout<< "is pulled: " << nodeInfoVecs.node_upper_selection_pull[i] << std::endl;
		std::cout<< "z: " << nodeInfoVecs.node_loc_z[i] << std::endl;
	}*/
	try{
		//try only counting collagen
		extensionParams.averageUpperStrain = (thrust::transform_reduce(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_is_collagen.begin(),
					nodeInfoVecs.node_upper_selection_pull.begin(),
					nodeInfoVecs.node_loc_z.begin())),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_is_collagen.begin(),
					nodeInfoVecs.node_upper_selection_pull.begin(),
					nodeInfoVecs.node_loc_z.begin())) + generalParams.max_node_count,
			functor_strain(generalParams.max_node_count, extensionParams.originalNetworkLength),
				0.0,
			thrust::plus<double>())) / generalParams.numUpperStrainNodes_collagen;
			

			extensionParams.averageLowerStrain = (thrust::transform_reduce(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.node_is_collagen.begin(),
						nodeInfoVecs.node_lower_selection_pull.begin(),
						nodeInfoVecs.node_loc_z.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.node_is_collagen.begin(),
						nodeInfoVecs.node_lower_selection_pull.begin(),
						nodeInfoVecs.node_loc_z.begin())) + generalParams.max_node_count,
				functor_strain(generalParams.max_node_count, extensionParams.originalNetworkLength),
					0.0,
				thrust::plus<double>())) / generalParams.numLowerStrainNodes_collagen;

	if (generalParams.iterationCounter == 1) {
		extensionParams.originAverageUpperStrain = extensionParams.averageUpperStrain;
		extensionParams.originAverageLowerStrain = extensionParams.averageLowerStrain;
	}

	//Apply External Force. 
	//Currently, we apply forces to all nodes withing a range of the average hight of those chosen for the upper/lower section. 
	//We always apply force to the collagen though. maybe remove that feature? currently collagen is allowed a window of 2micron, elastin 0.5
	thrust::counting_iterator<unsigned> indexBeginA(0);

	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				indexBeginA,
				nodeInfoVecs.node_loc_z.begin(),
				nodeInfoVecs.is_node_fixed.begin(),
				nodeInfoVecs.node_is_collagen.begin(),
				nodeInfoVecs.node_upper_selection_pull.begin(),
				nodeInfoVecs.node_lower_selection_pull.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				indexBeginA,
				nodeInfoVecs.node_loc_z.begin(),
				nodeInfoVecs.is_node_fixed.begin(),
				nodeInfoVecs.node_is_collagen.begin(),
				nodeInfoVecs.node_upper_selection_pull.begin(),
				nodeInfoVecs.node_lower_selection_pull.begin())) + generalParams.max_node_count,
		functor_external_force(
			thrust::raw_pointer_cast(nodeInfoVecs.is_node_fixed.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_force_x.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_force_y.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_force_z.data()),

			generalParams.magnitudeForce,
			extensionParams.originalNetworkLength,
			extensionParams.strain_proportion_end_sim,
			extensionParams.averageLowerStrain,
			extensionParams.averageUpperStrain));

		}
		catch(int e){std::cout<<"test"<< e <<std::flush;}
  };
