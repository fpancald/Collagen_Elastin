#include "system_structures.h"
#include "system.h"
#include "external_force.h"
#include "functor_strain.h"
#include "functor_external_force.h"


void external_force(
	NodeInfoVecs& nodeInfoVecs,
	GeneralParams& generalParams,
	ExtensionParams& extensionParams,
	DomainParams& domainParams){

    thrust::counting_iterator<unsigned> index_begin_upper(0);
		thrust::counting_iterator<unsigned> index_begin_lower(0);

		extensionParams.averageUpperStrain = (thrust::transform_reduce(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					index_begin_upper,
					nodeInfoVecs.node_upper_selection_pull.begin(),
					nodeInfoVecs.node_loc_z.begin())),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					index_begin_upper,
					nodeInfoVecs.node_upper_selection_pull.begin(),
					nodeInfoVecs.node_loc_z.begin())) + generalParams.max_node_count,
			functor_strain(generalParams.max_node_count, extensionParams.originalNetworkLength),
				0.0,
			thrust::plus<double>())) / generalParams.numUpperStrainNodes;

			extensionParams.averageLowerStrain = (thrust::transform_reduce(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						index_begin_lower,
						nodeInfoVecs.node_lower_selection_pull.begin(),
						nodeInfoVecs.node_loc_z.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						index_begin_lower,
						nodeInfoVecs.node_lower_selection_pull.begin(),
						nodeInfoVecs.node_loc_z.begin())) + generalParams.max_node_count,
				functor_strain(generalParams.max_node_count, extensionParams.originalNetworkLength),
					0.0,
				thrust::plus<double>())) / generalParams.numLowerStrainNodes;

	if (generalParams.iterationCounter == 1) {
		extensionParams.originAverageUpperStrain = extensionParams.averageUpperStrain;
		extensionParams.originAverageLowerStrain = extensionParams.averageLowerStrain;
	}


 
	//Apply External Force
	thrust::counting_iterator<unsigned> indexBeginA(0);

	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				indexBeginA,
				nodeInfoVecs.node_loc_z.begin(),
				nodeInfoVecs.is_node_fixed.begin(),
				nodeInfoVecs.node_upper_selection_pull.begin(),
				nodeInfoVecs.node_lower_selection_pull.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				indexBeginA,
				nodeInfoVecs.node_loc_z.begin(),
				nodeInfoVecs.is_node_fixed.begin(),
				nodeInfoVecs.node_upper_selection_pull.begin(),
				nodeInfoVecs.node_lower_selection_pull.begin())) + generalParams.max_node_count,
		functor_external_force(
			thrust::raw_pointer_cast(nodeInfoVecs.is_node_fixed.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_force_x.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_force_y.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_force_z.data()),

			generalParams.magnitudeForce,
			extensionParams.originalNetworkLength,
			extensionParams.strain_proportion_end_sim,
			extensionParams.averageLowerStrain,
			extensionParams.averageUpperStrain));


  };
