#include "system.h"
#include "system_structures.h"
#include "WLC_Force.h"
#include "functor_wlc.h"

/*
the structure of global_length_zero_index is
0  1  2  3
4  5  6  7
8  9  10 11
12 13 14 15 for a 4 node system.
index/4 = row,
index%4 = col. If you apply force to column node always or row node always then
each thread will apply opposing forces to springs.
if you decide to apply force to column instead of rows, you'll need sign change
LengthZero_value is symmetric, so values line up correctly.
*/

void calc_spring_force(
	NodeInfoVecs& nodeInfoVecs,
	EdgeInfoVecs& edgeInfoVecs,
	GeneralParams& generalParams) {


	thrust::counting_iterator<unsigned> startEdgeIter(0);

	//
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(startEdgeIter,
								nodeInfoVecs.is_node_fixed.begin() )),
		thrust::make_zip_iterator(
			thrust::make_tuple(startEdgeIter,
								nodeInfoVecs.is_node_fixed.begin() )) + generalParams.max_node_count,
		functor_collagen_elastin(
			thrust::raw_pointer_cast(nodeInfoVecs.node_loc_x.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_loc_y.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_loc_z.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_force_x.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_force_y.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_force_z.data()),

 			edgeInfoVecs.collagen_spring_constant,
			edgeInfoVecs.kB,
			generalParams.persistence_len_monomer,
			edgeInfoVecs.CLM,
			edgeInfoVecs.temperature,
			generalParams.max_nbr_count,
			generalParams.max_node_count,
			generalParams.nummonfiberarea,

			thrust::raw_pointer_cast(edgeInfoVecs.global_length_zero.data()),
			thrust::raw_pointer_cast(edgeInfoVecs.global_neighbors.data()),
			thrust::raw_pointer_cast(edgeInfoVecs.global_isedge_collagen.data()),
			thrust::raw_pointer_cast(edgeInfoVecs.global_isedge_elastin.data()),
			thrust::raw_pointer_cast(edgeInfoVecs.current_node_edge_count_vec.data()),
			thrust::raw_pointer_cast(edgeInfoVecs.num_origin_nbr_per_node_vec.data()) ) );
};
