#include "system.h"
#include "system_structures.h"
#include "bending_spring.h"

#include "functor_misc.h"
#include "functor_torsion.h"


void calc_bending_spring_force(
	NodeInfoVecs& nodeInfoVecs,
	BendInfoVecs& bendInfoVecs,
	GeneralParams& generalParams)  {

const double PI = 3.14159265358979323846;
if (bendInfoVecs.total_bend_count>0) {

		thrust::counting_iterator<unsigned> startTorsionIter(0);
		thrust::counting_iterator<unsigned> endTorsionIter(bendInfoVecs.total_bend_count);

		//for_each guarrantees order. This is needed for iter count and saving to torsion force vectors.
		thrust::for_each(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					startTorsionIter,
					bendInfoVecs.leftIndex.begin(),
					bendInfoVecs.centerIndex.begin(),
					bendInfoVecs.rightIndex.begin(),
					bendInfoVecs.angleZero.begin())),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					endTorsionIter,
					bendInfoVecs.leftIndex.end(),
					bendInfoVecs.centerIndex.end(),
					bendInfoVecs.rightIndex.end(),
					bendInfoVecs.angleZero.end())),
			functor_torsion(
				thrust::raw_pointer_cast(nodeInfoVecs.node_loc_x.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_loc_y.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_loc_z.data()),
				thrust::raw_pointer_cast(bendInfoVecs.forceX.data()),
				thrust::raw_pointer_cast(bendInfoVecs.forceY.data()),
				thrust::raw_pointer_cast(bendInfoVecs.forceZ.data()),

				thrust::raw_pointer_cast(nodeInfoVecs.is_node_fixed.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_is_collagen.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_is_elastin.data()),

				bendInfoVecs.bend_stiffness_collagen,
				bendInfoVecs.bend_stiffness_elastin,
				generalParams.max_node_count,
				bendInfoVecs.total_bend_count,
				PI));

		//reduce by key to get forces.Notice leftIndex is 1/3rd the length of torsion.forceX
		//this vector will be sorted each iteration, so it needs to be recopied.
		thrust::copy(bendInfoVecs.leftIndex.begin(), bendInfoVecs.leftIndex.end(), bendInfoVecs.tempTorIndices.begin());
		thrust::copy(bendInfoVecs.centerIndex.begin(), bendInfoVecs.centerIndex.end(), bendInfoVecs.tempTorIndices.begin() + bendInfoVecs.total_bend_count);
		thrust::copy(bendInfoVecs.rightIndex.begin(), bendInfoVecs.rightIndex.end(), bendInfoVecs.tempTorIndices.begin() + 2 * bendInfoVecs.total_bend_count);


		//key, then value. Each vector returns sorted
		thrust::sort_by_key(bendInfoVecs.tempTorIndices.begin(), bendInfoVecs.tempTorIndices.begin() + 3 * bendInfoVecs.total_bend_count,
			thrust::make_zip_iterator(
				thrust::make_tuple(
					bendInfoVecs.forceX.begin(),
					bendInfoVecs.forceY.begin(),
					bendInfoVecs.forceZ.begin())), thrust::less<unsigned>());


		thrust::fill(bendInfoVecs.tempForceX.begin(), bendInfoVecs.tempForceX.end(), 0);
		thrust::fill(bendInfoVecs.tempForceY.begin(), bendInfoVecs.tempForceY.end(), 0);
		thrust::fill(bendInfoVecs.tempForceZ.begin(), bendInfoVecs.tempForceZ.end(), 0);
		thrust::fill(bendInfoVecs.reducedIds.begin(), bendInfoVecs.reducedIds.end(), 0);

		unsigned endKey = thrust::get<0>(
			thrust::reduce_by_key(
				bendInfoVecs.tempTorIndices.begin(),
				bendInfoVecs.tempTorIndices.begin() + 3*bendInfoVecs.total_bend_count,
			thrust::make_zip_iterator(
				thrust::make_tuple(
					bendInfoVecs.forceX.begin(),
					bendInfoVecs.forceY.begin(),
					bendInfoVecs.forceZ.begin())),
			bendInfoVecs.reducedIds.begin(),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					bendInfoVecs.tempForceX.begin(),
					bendInfoVecs.tempForceY.begin(),
					bendInfoVecs.tempForceZ.begin())),
			thrust::equal_to<unsigned>(), CVec3Add())) - bendInfoVecs.reducedIds.begin();//binary_pred, binary_op


		thrust::for_each(
			thrust::make_zip_iterator(//1st begin
				thrust::make_tuple(
					bendInfoVecs.reducedIds.begin(),
					bendInfoVecs.tempForceX.begin(),
					bendInfoVecs.tempForceY.begin(),
					bendInfoVecs.tempForceZ.begin())),
			thrust::make_zip_iterator(//1st end
				thrust::make_tuple(
					bendInfoVecs.reducedIds.begin(),
					bendInfoVecs.tempForceX.begin(),
					bendInfoVecs.tempForceY.begin(),
					bendInfoVecs.tempForceZ.begin())) + endKey,
			functor_add_UCVec3_CVec3(
				generalParams.max_node_count,
				thrust::raw_pointer_cast(nodeInfoVecs.node_force_x.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_force_y.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_force_z.data())));

	}


}
