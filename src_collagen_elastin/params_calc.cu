#include "system_structures.h"
#include "functor_misc.h"

#include "system.h"
#include "Params_Calc.h"
#include "functor_calc_strain_params.h"


void Params_Calc(
    EdgeInfoVecs& edgeInfoVecs,
    NodeInfoVecs& nodeInfoVecs,
    GeneralParams& generalParams,
    PltInfoVecs& pltInfoVecs) {

		//count positive and negative strains for edges that are not added. If an edge is added, a zero is placed on that strain.
		//notice that each thread will count edges twice, so divide by two at the end
	
		thrust::fill(nodeInfoVecs.discretized_edges_strain.begin(), nodeInfoVecs.discretized_edges_strain.end(),0.0);
		thrust::fill(nodeInfoVecs.discretized_edges_alignment.begin(), nodeInfoVecs.discretized_edges_alignment.end(),0.0);	

		//copy current host information to device for strain calculation. 
		thrust::copy(nodeInfoVecs.hostEdgeLeft.begin(),
			nodeInfoVecs.hostEdgeLeft.begin() + generalParams.currentEdgeCount,
			nodeInfoVecs.device_edge_left.begin());

		thrust::copy(nodeInfoVecs.hostEdgeRight.begin(),
			nodeInfoVecs.hostEdgeRight.begin() + generalParams.currentEdgeCount,
			nodeInfoVecs.device_edge_right.begin());

		thrust::transform(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.device_edge_left.begin(),
					nodeInfoVecs.device_edge_right.begin())),
					 
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.device_edge_left.begin(),
					nodeInfoVecs.device_edge_right.begin())) + generalParams.currentEdgeCount,
					
			//outputs discretized strain etc			
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.discretized_edges_strain.begin(),
					nodeInfoVecs.discretized_edges_alignment.begin())),
					
			functor_calc_strain_params(
				generalParams.originLinkCount,
				generalParams.originEdgeCount,
				generalParams.originNodeCount,
				generalParams.max_node_count,
				generalParams.max_nbr_count,
				thrust::raw_pointer_cast(nodeInfoVecs.node_loc_x.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_loc_y.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_loc_z.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.num_origin_nbr_per_node_vec.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.current_node_edge_count_vec.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.global_neighbors.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.global_length_zero.data()) ));
		
			thrust::transform(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.node_force_x.begin(),
						nodeInfoVecs.node_force_y.begin(),
						nodeInfoVecs.node_force_z.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.node_force_x.begin(),
						nodeInfoVecs.node_force_y.begin(),
						nodeInfoVecs.node_force_z.begin())) + generalParams.max_node_count,
				nodeInfoVecs.sum_forces_on_node.begin(),//save vector
				
                functor_norm());

			//platelets
			thrust::transform(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						pltInfoVecs.pltForceX.begin(),
						pltInfoVecs.pltForceY.begin(),
						pltInfoVecs.pltForceZ.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						pltInfoVecs.pltForceX.begin(),
						pltInfoVecs.pltForceY.begin(),
						pltInfoVecs.pltForceZ.begin())) + generalParams.maxPltCount,
				pltInfoVecs.sumForcesOnPlt.begin(),//save vector
				functor_norm());
};